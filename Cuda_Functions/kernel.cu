#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

const size_t device = 0;

int width = 64, height = 64;
float* pDevice;
size_t pitch;



__global__ void addKernel(float* pDevice, size_t pitch, int width, int height){
    for (int r = 0; r < height; r++) {
        float* row = (float*)((char*)pDevice + r * pitch);
        for (int c = 0; c < width; c++) {
            float* element = row[c];
        }
    }
}

int main()
{
    hipMallocPitch(&pDevice, &pitch, width * sizeof(float), height);

    hipError_t cudaStatus = hipSetDevice(device);
    if (cudaStatus != hipSuccess) {fprintf (stderr, "addWithCuda failed!"); return 1; }

    return 0;
}

