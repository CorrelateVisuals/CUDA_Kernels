#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

__global__ void getKernelBlockDimensions(int* array) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    array[i] = blockDim.x;
}

__global__ void getKernelLocalhreadIndex(int* array) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    array[i] = threadIdx.x;
}

__global__ void getKernelBlockIndex(int* array) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    array[i] = blockIdx.x;
}

__global__ void getKernelThreadIndex(int* array) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    array[i] = i;
}

void printResults(int* array, int arraySize, const std::string& source) {
    std::cout << source;
    for (int i = 0; i < arraySize; i++) {
        std::cout << array[i] << " ";
    }
    std::cout << std::endl;
}

int main() {
    const dim3 gridSizex = { 3, 1, 1 };
    const dim3 numThreads = { 4, 1, 1 };
    const int arraySize = gridSizex.x * numThreads.x;

    int* h_array_block_dims = new int[arraySize];
    int* h_array_local_thread = new int[arraySize];
    int* h_array_block_index = new int[arraySize];
    int* h_array_thread_index = new int[arraySize];

    int* d_array_block_dims;
    int* d_array_local_thread;
    int* d_array_block_index;
    int* d_array_thread_index;

    // Block Dimensions
    hipMalloc((void**)&d_array_block_dims, arraySize * sizeof(int));
    hipMemcpy(d_array_block_dims, h_array_block_dims, arraySize * sizeof(int), hipMemcpyHostToDevice);
    getKernelBlockDimensions << <gridSizex, numThreads >> > (d_array_block_dims);

    // Local Thread Index
    hipMalloc((void**)&d_array_local_thread, arraySize * sizeof(int));
    hipMemcpy(d_array_local_thread, h_array_local_thread, arraySize * sizeof(int), hipMemcpyHostToDevice);
    getKernelLocalhreadIndex << <gridSizex, numThreads >> > (d_array_local_thread);

    // Block Index
    hipMalloc((void**)&d_array_block_index, arraySize * sizeof(int));
    hipMemcpy(d_array_block_index, h_array_block_index, arraySize * sizeof(int), hipMemcpyHostToDevice);
    getKernelBlockIndex << <gridSizex, numThreads >> > (d_array_block_index);

    // Thread Index
    hipMalloc((void**)&d_array_thread_index, arraySize * sizeof(int));
    hipMemcpy(d_array_thread_index, h_array_thread_index, arraySize * sizeof(int), hipMemcpyHostToDevice);
    getKernelThreadIndex << <gridSizex, numThreads >> > (d_array_thread_index);

    hipMemcpy(h_array_block_dims, d_array_block_dims, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_array_local_thread, d_array_local_thread, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_array_block_index, d_array_block_index, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_array_thread_index, d_array_thread_index, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_array_block_dims);
    hipFree(d_array_local_thread);
    hipFree(d_array_block_index);
    hipFree(d_array_thread_index);

    hipDeviceSynchronize();

    printResults(h_array_block_dims, arraySize, "Kernel block dimensions: ");
    printResults(h_array_local_thread, arraySize, "Kernel local thread index: ");
    printResults(h_array_block_index, arraySize, "Kernel block index: ");
    printResults(h_array_thread_index, arraySize, "Kernel thread index: ");

    delete[] h_array_block_dims;
    delete[] h_array_local_thread;
    delete[] h_array_block_index;
    delete[] h_array_thread_index;

    return 0;
}
