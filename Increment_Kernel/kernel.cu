#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>

void incrementCPU(int* a, int b, int N) {
    for (int i = 0; i < N; i++) {
        a[i] = a[i] + 10;
    }
}

__global__ void incrementGPU(int* a, int b, int N) {
    int i = threadIdx.x;
    if (i < N) {
        a[i] = a[i] + b;
    };
}

void printResults(int* a, int N, const std::string& source) {
    std::cout << "Incremented on " << source << ": { ";
    for (int i = 0; i < N; i++) {
        std::cout << a[i] << " ";
    }
    std::cout << "}" << std::endl;
}

int main() {
    const int N = 10;
    int stepSize = 10;
    int h_initialValues[N] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9 };

    // CPU execution
    incrementCPU(h_initialValues, stepSize, N);
    printResults(h_initialValues, N, "CPU");

    // GPU execution
    dim3 gridSize(1, 1, 1);
    dim3 blockSize(N, 1, 1);
    int* d_initialValues;

    hipMalloc((void**)&d_initialValues, N * sizeof(int));
    hipMemcpy(d_initialValues, h_initialValues, N * sizeof(int), hipMemcpyHostToDevice);

    incrementGPU << < gridSize, blockSize >> > (d_initialValues, stepSize, N);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_initialValues);
        return 1;
    }

    hipMemcpy(h_initialValues, d_initialValues, N * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_initialValues);

    printResults(h_initialValues, N, "GPU");

    return 0;
}
