﻿#include "hip/hip_runtime.h"

#include <iostream>

int stepSize = 10;

void incrementCPU(int* a, int arraySize) {
    for (size_t i = 0; i < arraySize; i++) {
        a[i] += stepSize;
    }
}

__global__ void incrementGPU(int* a, int arraySize) {
    size_t i = threadIdx.x;
    if (i < arraySize) {
        a[i] += stepSize;
    };
}

void printResults(int* a, int arraySize, const std::string& source) {
    std::cout << "Incremented on " << source << ": { ";
    for (size_t i = 0; i < arraySize; i++) {
        std::cout << a[i] << " ";
    }
    std::cout << "}" << std::endl;
}

int main() {
    const int arraySize = 10;
    int h_initialValues[arraySize] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9 };

    // CPU execution
    incrementCPU(h_initialValues, arraySize);
    printResults(h_initialValues, arraySize, "CPU");

    // GPU execution
    dim3 gridSize(1, 1, 1);
    dim3 blockSize(arraySize, 1, 1);
    int* d_initialValues = nullptr;

    hipMalloc((void**)&d_initialValues, arraySize * sizeof(int));
    hipMemcpy(d_initialValues, h_initialValues, arraySize * sizeof(int), hipMemcpyHostToDevice);

    incrementGPU<<< gridSize, blockSize >>>(d_initialValues, arraySize);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_initialValues);
        return 1;
    }

    hipMemcpy(h_initialValues, d_initialValues, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_initialValues);

    printResults(h_initialValues, arraySize, "GPU");

    return 0;
}
