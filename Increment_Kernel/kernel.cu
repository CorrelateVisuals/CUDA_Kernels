﻿#include "hip/hip_runtime.h"

#include <iostream>

void incrementCPU(int* a, int arraySize) {
    for (int i = 0; i < arraySize; i++) {
        a[i] = a[i] + 10;
    }
}

__global__ void incrementGPU(int* a, int arraySize) {
    int i = threadIdx.x;
    int stepSize = 10;
    if (i < arraySize) {
        a[i] = a[i] + stepSize;
    };
}

void printResults(int* a, int arraySize, const std::string& source) {
    std::cout << "Incremented on " << source << ": { ";
    for (int i = 0; i < arraySize; i++) {
        std::cout << a[i] << " ";
    }
    std::cout << "}" << std::endl;
}

int main() {
    const int arraySize = 10;
    int h_initialValues[arraySize] = { 0, 1, 2, 3, 4, 5, 6, 7, 8, 9 };

    // CPU execution
    incrementCPU(h_initialValues, arraySize);
    printResults(h_initialValues, arraySize, "CPU");

    // GPU execution
    dim3 gridSize(1, 1, 1);
    dim3 blockSize(arraySize, 1, 1);
    int* d_initialValues;

    hipMalloc((void**)&d_initialValues, arraySize * sizeof(int));
    hipMemcpy(d_initialValues, h_initialValues, arraySize * sizeof(int), hipMemcpyHostToDevice);

    incrementGPU<<< gridSize, blockSize >>>(d_initialValues, arraySize);

    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        hipFree(d_initialValues);
        return 1;
    }

    hipMemcpy(h_initialValues, d_initialValues, arraySize * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_initialValues);

    printResults(h_initialValues, arraySize, "GPU");

    return 0;
}
