#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <string>
#include <cassert>

#include "STB_Image_Load.h"
#include "STB_Image_Write.h"

struct alignas(16) Pixel {
    unsigned char r, g, b, a;
};
const int pixelSizeBytes = sizeof(Pixel);
const int channels = pixelSizeBytes / sizeof(int);
const unsigned char maxIntensity = 255;

const struct alignas(16) greyScaleRGB {
    const float r = 0.2126f;
    const float g = 0.7152f;
    const float b = 0.0722f;
};

const int blockDimension = 32;

__global__ void threadsOnImage(unsigned char* imageRGBA) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t id = y * blockDim.x * gridDim.x + x;

    Pixel* pPixel = (Pixel*)&imageRGBA[id * channels];
    greyScaleRGB weight;
    unsigned char pixelValue = (unsigned char)(pPixel->r * weight.r + pPixel->g * weight.g + pPixel->b * weight.b);
    unsigned char offset = 50;
    int a_value = pixelValue + offset;
    pPixel->a = static_cast<unsigned char>(a_value < 0 ? 0 : (a_value > maxIntensity ? maxIntensity : a_value));

    int intensityX = threadIdx.x % blockDim.x + blockIdx.x;
    int intensityY = threadIdx.y % blockDim.y + blockIdx.y; 
    int intensity = intensityY + intensityX;
    pPixel->r = static_cast<unsigned char>( intensity );
    pPixel->g = static_cast<unsigned char>(x);
    pPixel->b = static_cast<unsigned char>(y);
}

__global__ void imageToGreyscale(unsigned char* imageRGBA) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t id = y * blockDim.x * gridDim.x + x;

    Pixel* pPixel = (Pixel*)&imageRGBA[id * channels];
    greyScaleRGB weight;
    unsigned char pixelValue = (unsigned char)(pPixel->r * weight.r + pPixel->g * weight.g + pPixel->b * weight.b);
    pPixel->r = pixelValue;
    pPixel->g = pixelValue;
    pPixel->b = pixelValue;
    pPixel->a = maxIntensity;
}

void copyDataAndWriteToDisk(unsigned char* pImageDataGPU, unsigned char* imageData, int width, int height, int channels, const std::string& fileNameOut, int strideBytes, const std::string& suffix) {
    std::cout << "Copy data from GPU ... ";
    assert(hipMemcpy(imageData, pImageDataGPU, width * height * channels, hipMemcpyDeviceToHost) == hipSuccess);
    std::cout << "DONE" << std::endl;

    std::cout << "Writing png to disk ... ";
    std::string baseFileName = fileNameOut.substr(0, fileNameOut.find_last_of("."));
    std::string newFileName = baseFileName + suffix;
    stbi_write_png(newFileName.c_str(), width, height, channels, imageData, strideBytes);
    std::cout << "Done " << fileNameOut << " saved" << std::endl;
}

int main(int argc, char** argv) {
    if (argc < 2) { std::cout << "Usage: Image Color Manipulation <filename>" << std::endl; return -1; }

    int width, height, componentCount;
    std::cout << "Loading png file ... ";
    unsigned char* imageData = stbi_load(argv[1], &width, &height, &componentCount, channels);
    if (!imageData) { std::cout << "Failed to open \"" << argv[1] << "\"";  return -1; }
    std::cout << "Done" << std::endl;

    std::cout << "Copy data to GPU ... ";
    if (width % 32 || height % 32) { std::cout << "Width or height is not devidible by 32; leaked memory of imageData"; return -1; }
    unsigned char* pImageDataGPU = nullptr;
    assert(hipMalloc(&pImageDataGPU, width * height * channels) == hipSuccess);
    assert(hipMemcpy(pImageDataGPU, imageData, width * height * channels, hipMemcpyHostToDevice) == hipSuccess);

    dim3 blockSize(blockDimension, blockDimension, 1);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    std::string fileNameOut = argv[1];
    int strideBytes = channels * width;

    std::cout << "Running CUDA kernel ... ";
    imageToGreyscale<<<gridSize, blockSize>>>(pImageDataGPU);
    std::cout << "DONE" << std::endl;
    copyDataAndWriteToDisk(pImageDataGPU, imageData, width, height, channels, fileNameOut, strideBytes, "_grey.jpg");

    std::cout << "Running CUDA kernel ... ";
    threadsOnImage<<<gridSize, blockSize>>>(pImageDataGPU);
    std::cout << "DONE" << std::endl;
    copyDataAndWriteToDisk(pImageDataGPU, imageData, width, height, channels, fileNameOut, strideBytes, "_threads.jpg");

    hipFree(pImageDataGPU);
    stbi_image_free(imageData);

    return 0;
}
